#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <ctype.h>
#include "sysstuff.h"
#include "typedefs.h"
#include "macros.h"
#include "vec.h"
#include "pbc.h"
#include "xvgr.h"
#include "gromacs/fileio/futil.h"
#include "gromacs/commandline/pargs.h"
#include "gromacs/fileio/tpxio.h"
#include "gromacs/fileio/trxio.h"
#include "physics.h"
#include "index.h"
#include "gromacs/utility/smalloc.h"
#include "calcgrid.h"
#include "nrnb.h"
#include "coulomb.h"
#include "../gstat.h"
#include "gromacs/fileio/matio.h"
#include "../gmx_ana.h"
#include "../hyperpol.h"
#include "names.h"
#include "hip/hip_runtime.h"
#include "gromacs/legacyheaders/gmx_fatal.h"
#include "gromacs/gmxlib/cuda_tools/cudautils.cuh"


#define block_size 1024  // number of threads per block for kernel launching

__global__ 
void kernel_fade(t_pbc *pbc,real beta_i,real *beta,rvec x_i,rvec *x,real rmax2,rvec qvec,real *result,int n,int N,
                 real fade,real inv_width) {

  __shared__ real buffer[block_size];
  int j,k;
  rvec dx;
  real r,r2,temp=0;

  // Obtain the thread ID
  j=threadIdx.x+blockIdx.x*blockDim.x;

  // Prevent the unnecessary threads
  if (j>=N)  return;

  // Initialize the shared memory in each Block
  //buffer[threadIdx.x]=0.0;

  // Calculate vector from the ith to jth molecule
  dx[XX]=(x[j][XX]-x_i[XX]);
  dx[YY]=(x[j][YY]-x_i[YY]);
  dx[ZZ]=(x[j][ZZ]-x_i[ZZ]);

  // Apply PBC condition
  dx[XX]-=rintf(dx[XX]/pbc->box[XX][XX])*pbc->box[XX][XX];
  dx[YY]-=rintf(dx[YY]/pbc->box[YY][YY])*pbc->box[YY][YY];
  dx[ZZ]-=rintf(dx[ZZ]/pbc->box[ZZ][ZZ])*pbc->box[ZZ][ZZ];

  // Distance sqaure between ith and jth
  r2=(dx[XX]*dx[XX]+dx[YY]*dx[YY]+dx[ZZ]*dx[ZZ]);
  
  if (r2>0 && r2<=rmax2) {
    r=sqrt(r2);

    if (r <= fade) {
      temp=beta_i*beta[j]*cos(qvec[XX]*dx[XX]+qvec[YY]*dx[YY]+qvec[ZZ]*dx[ZZ]);
    }
    else {
       temp=beta_i*beta[j]*cos(qvec[XX]*dx[XX]+qvec[YY]*dx[YY]+qvec[ZZ]*dx[ZZ])*sqr(cos((r-fade)*inv_width));
    }
  }

  // Save each element to the shared memory
  buffer[threadIdx.x]=temp; 

  // Synchronize all the threads
  __syncthreads(); 

  // Summation Reduction
  k=blockDim.x*0.5;

  while (k!=0) {
    if (threadIdx.x<k) {
      buffer[threadIdx.x]+=buffer[threadIdx.x+k];
    }
    __syncthreads();
    k*=0.5;
  }

  // Return the first element which contains the sum of the block
  if (threadIdx.x==0) {
    result[blockIdx.x]=buffer[0];
    //printf("result[%d][%d] = %f\n",blockIdx.x,blockIdx.y,buffer[0]);
  }
}


extern void double_sum_fade(t_pbc *pbc,real *beta,rvec *x,int n,real rmax2,rvec *qvec,int nbinq,real fade,real inv_width,real *temp_method) {

  int i,j,q,N=0,grid_size=0;
  t_pbc *pbc_d=NULL;
  rvec *x_d=NULL,*qvec_d=NULL;
  real *beta_d=NULL,*result=NULL,*result_d=NULL,*Sum=NULL;
  //hipError_t status;

  // Determine grid_size of the kernel
  grid_size=(int)(n+block_size-1)/block_size;

  // Determine total number of threads
  N=block_size*grid_size;

  // Allocate Host Memory
  result=(real *)calloc(grid_size,sizeof(real));
  Sum=(real *)calloc(nbinq,sizeof(real));  

  // Allocate Device Memory
  hipMalloc(&pbc_d,sizeof(t_pbc));
  hipMalloc(&x_d,sizeof(rvec)*n);
  hipMalloc(&qvec_d,sizeof(rvec)*nbinq);
  hipMalloc(&beta_d,sizeof(real)*n);
  hipMalloc(&result_d,sizeof(real)*grid_size);

  // Copy memory from Host to Device
  hipMemcpy(pbc_d,pbc,sizeof(t_pbc),hipMemcpyHostToDevice);
  hipMemcpy(x_d,x,sizeof(rvec)*n,hipMemcpyHostToDevice);
  hipMemcpy(qvec_d,qvec,sizeof(rvec)*nbinq,hipMemcpyHostToDevice);
  hipMemcpy(beta_d,beta,sizeof(real)*n,hipMemcpyHostToDevice);

  // Calculate individual water in series
  for (i=0;i<n;i++) {  
    for (q=0;q<nbinq;q++) {
      // Launch kernel with N threads
      kernel_fade<<<grid_size,block_size>>>(pbc_d,beta[i],beta_d,x_d[i],x_d,rmax2,qvec_d[q],result_d,n,N,fade,inv_width);
      //status=hipGetLastError();
      //printf("kernel: %s\n",hipGetErrorString(status));

      // Copy result back from Device to Host 
      //status=
      hipMemcpy(result,result_d,sizeof(real)*grid_size,hipMemcpyDeviceToHost);
      //printf("copy D2H %s\n",hipGetErrorString(status)); 

      // Sum over the individual sum in each block
      for (j=0;j<grid_size;j++) {      
        Sum[q]+=result[j];
        //printf("block = %d, q = %d, result = %f\n",j,q,result[j+grid_size*q]);
      }
    }
  }

  // Send the number back
  for (q=0;q<nbinq;q++) {
    temp_method[q]=Sum[q]*0.5;
  }

  // Free Device Memory
  hipFree(pbc_d);
  hipFree(x_d);   
  hipFree(beta_d); 
  hipFree(result_d);

  // Free Host Memory
  free(result);

  return;
}


